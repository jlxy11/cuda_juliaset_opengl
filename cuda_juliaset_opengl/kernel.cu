#include "hip/hip_runtime.h"
﻿/*CPU*/
//#include <GL/glut.h>
//#include <cmath>
//#include <chrono>
//#include<iostream>
//
//std::chrono::high_resolution_clock::time_point previousTime;
//int frameCount = 0;
//float frameRate = 0.0;
//
//
//const int n = 800;
//const int WIDTH = n;
//const int HEIGHT = n;
//
//const int MAX_ITER = 100;
//const float BOUNDARY = 2.0;
//
//const float MIN_REAL = -2.0;
//const float MAX_REAL = 2.0;
//const float MIN_IMAG = -2.0;
//const float MAX_IMAG = 2.0;
//
//void display()
//{
//    glClear(GL_COLOR_BUFFER_BIT);
//    glLoadIdentity();
//
//    /*double dx = 2.0 / WIDTH;
//    double dy = 2.0 / HEIGHT;*/
//
//    float dx = (MAX_REAL - MIN_REAL) / WIDTH;
//    float dy = (MAX_IMAG - MIN_IMAG) / HEIGHT;
//
//    float t = glutGet(GLUT_ELAPSED_TIME) * 0.0002; // 获取时间，单位为秒
//    //double c_real = -0.8;
//    //double c_imag = 0.2 * cos(t);
//    float c_real = 0.7885 * sin(t);
//    float c_imag = 0.7885 * cos(t);
//    glBegin(GL_POINTS);
//    for (int i = 0; i < WIDTH; ++i)
//    {
//        for (int j = 0; j < HEIGHT; ++j)
//        {
//
//            //double x = (-1.0 + i * dx);
//            //double y = (-1.0 + j * dy);
//            float x = MIN_REAL + i * dx;
//            float y = MIN_IMAG + j * dy;
//
//            float zx = x;
//            float zy = y;
//
//            int iter = 0;
//            while (iter < MAX_ITER && zx * zx + zy * zy < BOUNDARY * BOUNDARY)
//            {
//                float tmp = zx * zx - zy * zy + c_real;
//                zy = 2.0 * zx * zy + c_imag;
//                zx = tmp;
//                ++iter;
//            }
//
//            // 根据迭代次数绘制不同颜色的点
//            //float color = 1.0f - static_cast<float>(iter) / MAX_ITER;
//            float colorRED = (-130.0 * sinf(0.045 * (iter + 13.3)) + 160.0) / 255.0;
//            float colorGREEN = (-80.0 * sinf(0.045 * (iter + 0.0)) + 40.0) / 225.0;
//            float colorBLUE = (60.0 * sinf(0.1 * (iter - 29.3)) + 80.0) / 225.0;
//
//            if (colorRED > 1.0)
//            {
//                colorRED = 1.0;
//            }
//            else if (colorRED < 0.0)
//            {
//                colorRED = 0.0;
//            }
//
//            if (colorGREEN > 1.0)
//            {
//                colorGREEN = 1.0;
//            }
//            else if (colorGREEN < 0.0)
//            {
//                colorGREEN = 0.0;
//            }
//
//            if (colorBLUE > 1.0)
//            {
//                colorBLUE = 1.0;
//            }
//            else if (colorBLUE < 0.0)
//            {
//                colorBLUE = 0.0;
//            }
//
//            //if (iter == MAX_ITER)
//            //{
//            //    colorRED = 0.0;
//            //    colorGREEN = 0.0;
//            //    colorBLUE = 0.0;
//
//            //}
//            //
//            glColor3f(colorRED, colorGREEN, colorBLUE);
//            //glColor3f(color, color, color);
//            glVertex2d(x, y);
//            //A*SIN((I-D)*K)+C
//
//        }
//    }
//
//    glEnd();
//
//    glFlush();
//    glutSwapBuffers();
//    
//    std::chrono::high_resolution_clock::time_point currentTime = std::chrono::high_resolution_clock::now();
//    double elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(currentTime - previousTime).count() / 1000.0;
//    frameCount++;
//    if (elapsedTime >= 1.0) {
//        frameRate = frameCount / elapsedTime;
//        frameCount = 0;
//        previousTime = currentTime;
//    }
//
//    // 在控制台输出帧率
//    std::cout << "Frame rate: " << frameRate << " fps" << std::endl;
//}
//
//void reshape(int w, int h)
//{
//    glViewport(0, 0, w, h);
//    glMatrixMode(GL_PROJECTION);
//    glLoadIdentity();
//    //gluOrtho2D(-1, 1, -1, 1);
//    gluOrtho2D(MIN_REAL, MAX_REAL, MIN_IMAG, MAX_IMAG);
//    glMatrixMode(GL_MODELVIEW);
//}
//
//int main(int argc, char** argv)
//{
//    previousTime = std::chrono::high_resolution_clock::now();
//    glutInit(&argc, argv);
//    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
//    glutInitWindowSize(WIDTH, HEIGHT);
//    glutCreateWindow("Julia Set Visualization");
//    glutDisplayFunc(display);
//    glutReshapeFunc(reshape);
//    glutIdleFunc(display); // 设置空闲时刷新画面
//    glutMainLoop();
//    return 0;
//}

/*CUDA_slow*/
//#include <GL/glut.h>
//#include <stdio.h>
//#include <stdlib.h>
//#include <math.h>
//#include <hip/hip_runtime.h>
//#include ""
//#include<chrono>
//#include<iostream>
//
//std::chrono::high_resolution_clock::time_point previousTime;
//int frameCount = 0;
//double frameRate = 0.0;
//
//
//const int n = 800;
//const int WIDTH = n;
//const int HEIGHT = n;
//
//const int MAX_ITER = 100;
//const double BOUNDARY = 2.0;
//
//const double MIN_REAL = -2.0;
//const double MAX_REAL = 2.0;
//const double MIN_IMAG = -2.0;
//const double MAX_IMAG = 2.0;
//
//__global__ void juliaSetKernel(float* output, float* xpix, float* ypix, float t)
//{
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    int idy = blockIdx.y * blockDim.y + threadIdx.y;
//    int offset = idy * WIDTH + idx;
//    if (offset < WIDTH * HEIGHT)
//    {
//        float x = (idx - WIDTH / 2) / (float)WIDTH * 4;
//        float y = (idy - HEIGHT / 2) / (float)HEIGHT * 4;
//        //float c_real = 0.0;
//        //float c_imag = 0.67;
//        double c_real = 0.7885 * sin(t);
//        double c_imag = 0.7885 * cos(t);
//        float z_real = x;
//        float z_imag = y;
//
//        int iter;
//        for (iter = 0; iter < MAX_ITER; iter++) {
//            float z_real2 = z_real * z_real;
//            float z_imag2 = z_imag * z_imag;
//            if (z_real2 + z_imag2 > 4.0)
//                break;
//
//            float tmp = z_real2 - z_imag2 + c_real;
//            z_imag = 2.0 * z_real * z_imag + c_imag;
//            z_real = tmp;
//        }
//
//        output[offset] = iter;
//        xpix[offset] = x;
//        ypix[offset] = y;
//    }
//
//    
//}
//
//void juliaSet(float* output, float* xpix, float* ypix, float t)
//{
//    float* dev_output;
//    float* dev_xpix;
//    float* dev_ypix;
//
//    hipMalloc((void**)&dev_output, WIDTH * HEIGHT * sizeof(float));
//    hipMalloc((void**)&dev_xpix, WIDTH * HEIGHT * sizeof(float));
//    hipMalloc((void**)&dev_ypix, WIDTH * HEIGHT * sizeof(float));
//
//    dim3 threadsPerBlock(16, 16);
//    dim3 numBlocks(WIDTH / threadsPerBlock.x, HEIGHT / threadsPerBlock.y);
//
//    juliaSetKernel << <numBlocks, threadsPerBlock >> > (dev_output,dev_xpix, dev_ypix, t);
//
//    hipMemcpy(output, dev_output, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
//    hipMemcpy(xpix, dev_xpix, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
//    hipMemcpy(ypix, dev_ypix, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
//
//    hipFree(dev_xpix);
//    hipFree(dev_ypix);
//    hipFree(dev_output);
//}
//
//void reshape(int w, int h)
//{
//    glViewport(0, 0, w, h);
//    glMatrixMode(GL_PROJECTION);
//    glLoadIdentity();
//    //gluOrtho2D(-1, 1, -1, 1);
//    gluOrtho2D(MIN_REAL, MAX_REAL, MIN_IMAG, MAX_IMAG);
//    glMatrixMode(GL_MODELVIEW);
//}
//
//void display()
//{
//    glClear(GL_COLOR_BUFFER_BIT);
//    glLoadIdentity();
//    double t = glutGet(GLUT_ELAPSED_TIME) * 0.0002; // 获取时间，单位为秒
//    float* output = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
//    float* x = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
//    float* y = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
//
//    juliaSet(output, x, y, t);
//
//    glBegin(GL_POINTS); // 开始绘制点
//
//    for (int i = 0; i < WIDTH * HEIGHT; i++)
//    {
//        float iter = output[i];
//
//        //float color = 1.0f - static_cast<float>(iter) / MAX_ITER;
//        //float color = 1.0f - static_cast<float>(iter) / MAX_ITER;
//        float colorRED = (-130.0 * sinf(0.045 * (iter + 13.3)) + 160.0) / 255.0;
//        float colorGREEN = (-80.0 * sinf(0.045 * (iter + 0.0)) + 40.0) / 225.0;
//        float colorBLUE = (60.0 * sinf(0.1 * (iter - 29.3)) + 80.0) / 225.0;
//
//        if (colorRED > 1.0)
//        {
//            colorRED = 1.0;
//        }
//        else if (colorRED < 0.0)
//        {
//            colorRED = 0.0;
//        }
//
//        if (colorGREEN > 1.0)
//        {
//            colorGREEN = 1.0;
//        }
//        else if (colorGREEN < 0.0)
//        {
//            colorGREEN = 0.0;
//        }
//
//        if (colorBLUE > 1.0)
//        {
//            colorBLUE = 1.0;
//        }
//        else if (colorBLUE < 0.0)
//        {
//            colorBLUE = 0.0;
//        }
//
//        glColor3f(colorRED, colorGREEN, colorBLUE);
//        //glColor3f(color, color, color);
//        glVertex2d(x[i], y[i]);
//    }
//    free(output);
//    free(x);
//    free(y);
//
//    glEnd(); // 结束绘制点
//
//    glFlush();
//    glutSwapBuffers();
//    // 计算帧率
//    std::chrono::high_resolution_clock::time_point currentTime = std::chrono::high_resolution_clock::now();
//    double elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(currentTime - previousTime).count() / 1000.0;
//    frameCount++;
//    if (elapsedTime >= 1.0) {
//        frameRate = frameCount / elapsedTime;
//        frameCount = 0;
//        previousTime = currentTime;
//    }
//
//    // 在控制台输出帧率
//    std::cout << "Frame rate: " << frameRate << " fps" << std::endl;
//
//}
//
//int main(int argc, char** argv)
//{
//    previousTime = std::chrono::high_resolution_clock::now();
//    glutInit(&argc, argv);
//    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
//    glutInitWindowSize(WIDTH, HEIGHT);
//    glutCreateWindow("Julia Set Visualization");
//    glutDisplayFunc(display);
//    glutReshapeFunc(reshape);
//    glutIdleFunc(display); // 设置空闲时刷新画面
//    glutMainLoop();
//    return 0;
//}

/*CUDA_static*/
//#include <stdio.h>
//#include <stdlib.h>
//#include <math.h>
//#include <hip/hip_runtime.h>
//#include ""
//
//#define WIDTH 800
//#define HEIGHT 800
//#define MAX_ITER 100
//
//
//__global__ void juliaSetKernel(float* output, float* xpix, float* ypix, float t)
//{
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    int idy = blockIdx.y * blockDim.y + threadIdx.y;
//    int offset = idy * WIDTH + idx;
//
//    float x = (idx - WIDTH / 2) / (float)WIDTH * 4;
//    float y = (idy - HEIGHT / 2) / (float)HEIGHT * 4;
//    float c_real = 0.7885 * sin(t);
//    float c_imag = 0.7885 * cos(t);
//    float z_real = x;
//    float z_imag = y;
//
//    int iter;
//    for (iter = 0; iter < MAX_ITER; iter++) {
//        float z_real2 = z_real * z_real;
//        float z_imag2 = z_imag * z_imag;
//        if (z_real2 + z_imag2 > 4.0)
//            break;
//
//        float tmp = z_real2 - z_imag2 + c_real;
//        z_imag = 2.0 * z_real * z_imag + c_imag;
//        z_real = tmp;
//    }
//
//    output[offset] = iter;
//    xpix[offset] = x;
//    ypix[offset] = y;
//}
//
//void juliaSet(float* output, float* xpix, float* ypix, float t)
//{
//    float* dev_output;
//    float* dev_xpix;
//    float* dev_ypix;
//
//    hipMalloc((void**)&dev_output, WIDTH * HEIGHT * sizeof(float));
//    hipMalloc((void**)&dev_xpix, WIDTH * HEIGHT * sizeof(float));
//    hipMalloc((void**)&dev_ypix, WIDTH * HEIGHT * sizeof(float));
//
//
//    dim3 threadsPerBlock(16, 16);
//    dim3 numBlocks(WIDTH / threadsPerBlock.x, HEIGHT / threadsPerBlock.y);
//
//    juliaSetKernel << <numBlocks, threadsPerBlock >> > (dev_output, dev_xpix, dev_ypix,t);
//
//    hipMemcpy(output, dev_output, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
//    hipMemcpy(xpix, dev_xpix, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
//    hipMemcpy(ypix, dev_ypix, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
//
//    hipFree(dev_output);
//    hipFree(dev_xpix);
//    hipFree(dev_ypix);
//
//}
//
//int main()
//{
//    float* output = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
//    float* x = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
//    float* y = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
//
//    float t = 1.0; // 当前时间
//    juliaSet(output, x, y, t);
//
//    // 输出存储迭代次数的数组
//    for (int i = 0; i < WIDTH * HEIGHT; i++) {
//        //printf("%f ", output[i]);
//        //printf("%f ", x[i],y[i]);
//        if (output[i] > 20.0)
//        {
//            printf("%f ", output[i]);
//        }
//    }
//    
//
//    free(output);
//    printf("done");
//    return 0;
//}

/*CUDA_last_version*/
#include <GL/glut.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include ""
#include<chrono>
#include<iostream>

std::chrono::high_resolution_clock::time_point previousTime;
int frameCount = 0;
float frameRate = 0.0;


const int n = 800;
const int WIDTH = n;
const int HEIGHT = n;

const int MAX_ITER = 100;
const float BOUNDARY = 2.0;

const float MIN_REAL = -2.0;
const float MAX_REAL = 2.0;
const float MIN_IMAG = -2.0;
const float MAX_IMAG = 2.0;


float* x = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
float* y = (float*)malloc(WIDTH * HEIGHT * sizeof(float));

void initdraw()
{
    for (int i = 0; i < WIDTH; ++i)
    {
        for (int j = 0; j < HEIGHT; ++j)
        {
            int offset = j * WIDTH + i;
            x[offset] = (i - WIDTH / 2) / (float)WIDTH * 4;
            y[offset] = (j - HEIGHT / 2) / (float)HEIGHT * 4;

        }
    }
}

__device__ int iterateJuliaSet(float z_real, float z_imag, float c_real, float c_imag)
{
    int iter;
    for (iter = 0; iter < MAX_ITER; iter++) {
        float z_real2 = z_real * z_real;
        float z_imag2 = z_imag * z_imag;
        if (z_real2 + z_imag2 > 4.0)
            break;

        float tmp = z_real2 - z_imag2 + c_real;
        z_imag = 2.0 * z_real * z_imag + c_imag;
        z_real = tmp;
    }
    return iter;
}

__global__ void juliaSetKernel(float* colorRED, float* colorGREEN, float* colorBLUE, float t)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = idy * WIDTH + idx;
    if (offset < WIDTH * HEIGHT)
    {
        float x = (idx - WIDTH / 2) / (float)WIDTH * 4;
        float y = (idy - HEIGHT / 2) / (float)HEIGHT * 4;
        float c_real = 0.7885 * sin(t);
        float c_imag = 0.7885 * cos(t);
        float z_real = x;
        float z_imag = y;

        int iter = iterateJuliaSet(z_real, z_imag, c_real, c_imag);
        //float color = 1.0f - static_cast<float>(iter) / MAX_ITER;
        //float color = 1.0f - static_cast<float>(iter) / MAX_ITER;
        colorRED[offset] = (-130.0 * sinf(0.045 * (iter + 13.3)) + 160.0) / 255.0;
        if (colorRED[offset] > 1.0)
        {
            colorRED[offset] = 1.0;
        }
        else if (colorRED[offset] < 0.0)
        {
            colorRED[offset] = 0.0;
        }

        colorGREEN[offset] = (-80.0 * sinf(0.045 * (iter + 0.0)) + 40.0) / 225.0;
        if (colorGREEN[offset] > 1.0)
        {
            colorGREEN[offset] = 1.0;
        }
        else if (colorGREEN[offset] < 0.0)
        {
            colorGREEN[offset] = 0.0;
        }

        colorBLUE[offset] = (60.0 * sinf(0.1 * (iter - 29.3)) + 80.0) / 225.0;
        if (colorBLUE[offset] > 1.0)
        {
            colorBLUE[offset] = 1.0;
        }
        else if (colorBLUE[offset] < 0.0)
        {
            colorBLUE[offset] = 0.0;
        }
    }
}

void juliaSet(float* colorRED, float* colorGREEN, float* colorBLUE, float t)
{
    float* dev_colorRED;
    float* dev_colorGREEN;
    float* dev_colorBLUE;

    hipMalloc((void**)&dev_colorRED, WIDTH * HEIGHT * sizeof(float));
    hipMalloc((void**)&dev_colorGREEN, WIDTH * HEIGHT * sizeof(float));
    hipMalloc((void**)&dev_colorBLUE, WIDTH * HEIGHT * sizeof(float));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(WIDTH / threadsPerBlock.x, HEIGHT / threadsPerBlock.y);

    juliaSetKernel << <numBlocks, threadsPerBlock >> > (dev_colorRED, dev_colorGREEN, dev_colorBLUE, t);

    hipMemcpy(colorRED, dev_colorRED, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(colorGREEN, dev_colorGREEN, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(colorBLUE, dev_colorBLUE, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(dev_colorRED);
    hipFree(dev_colorGREEN);
    hipFree(dev_colorBLUE);
}

void reshape(int w, int h)
{
    glViewport(0, 0, w, h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(MIN_REAL, MAX_REAL, MIN_IMAG, MAX_IMAG);
    glMatrixMode(GL_MODELVIEW);
}

void display()
{
    glClear(GL_COLOR_BUFFER_BIT);
    glLoadIdentity();
    double t = glutGet(GLUT_ELAPSED_TIME) * 0.0002; // 获取时间，单位为秒
    float* colorRED = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
    float* colorGREEN = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
    float* colorBLUE = (float*)malloc(WIDTH * HEIGHT * sizeof(float));

    juliaSet(colorRED, colorGREEN, colorBLUE, t);
    glBegin(GL_POINTS); // 开始绘制点
    
    for (int i = 0; i < WIDTH * HEIGHT; i++)
    {
        glColor3f(colorRED[i], colorGREEN[i], colorBLUE[i]);
        glVertex2d(x[i], y[i]);
    }
    free(colorRED);
    free(colorGREEN);
    free(colorBLUE);

    glEnd(); // 结束绘制点

    glFlush();
    glutSwapBuffers();
    // 计算帧率
    std::chrono::high_resolution_clock::time_point currentTime = std::chrono::high_resolution_clock::now();
    float elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(currentTime - previousTime).count() / 1000.0;
    frameCount++;
    if (elapsedTime >= 1.0) {
        frameRate = frameCount / elapsedTime;
        frameCount = 0;
        previousTime = currentTime;
    }

    // 在控制台输出帧率
    std::cout << "Frame rate: " << frameRate << " fps" << std::endl;

}

int main(int argc, char** argv)
{
    initdraw();
    previousTime = std::chrono::high_resolution_clock::now();
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("Julia Set Visualization");
    glutDisplayFunc(display);
    glutReshapeFunc(reshape);
    glutIdleFunc(display); // 设置空闲时刷新画面
    glutMainLoop();
    return 0;
}